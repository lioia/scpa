#include <iostream>

#include <hip/hip_runtime.h> // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h> // For CUDA SDK timers

int main(void) {
  std::cout << "Hello, world!" << std::endl;
  return 0;
}
